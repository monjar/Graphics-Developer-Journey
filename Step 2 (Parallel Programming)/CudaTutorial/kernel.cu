#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <math.h>
#include <chrono>

void addBasic(int n, float* x, float* y)
{

    for (int i = 0; i < n; i += 1)
        y[i] = x[i] + y[i];

}
// Kernel function to add the elements of two arrays
__global__
void add(int n, float* x, float* y)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
    
}

__global__
void addBlocks(int n, float* x, float* y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}



int main(void)
{
    int N = 100000000;
    float* x, * y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    std::cout << "Initializing... " << std::endl;
    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    std::cout << "Adding: " << std::endl;
    auto start1 = std::chrono::steady_clock::now();

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    addBlocks << <numBlocks, blockSize >> > (N, x, y);

    //addBasic(N, x, y);
    //add << <1, 256 >> > (N, x, y);
    hipDeviceSynchronize();

    auto end1 = std::chrono::steady_clock::now();

    std::cout << "Time 1: " << std::chrono::duration<double, std::milli>((end1 - start1)).count() << std::endl;
  

    hipFree(x);
    hipFree(y);

    return 0;
}